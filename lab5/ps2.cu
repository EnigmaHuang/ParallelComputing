
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void VecAdd(float* A, float* B, float* C, int N){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

int main(int argc, char** argv){
    srand(2634);
    int N = atoi(argv[1]);
    char* out = argv[2];

    hipEvent_t start, stop, fin;
    float dur_time;

    size_t size = N * sizeof(float);
    float* h_A;
    hipHostMalloc((void**)&h_A, size, hipHostMallocDefault);
    float* h_B;
    hipHostMalloc((void**)&h_B, size, hipHostMallocDefault);
    float* h_C;
    hipHostMalloc((void**)&h_C, size, hipHostMallocDefault);
    float* h_D;
    hipHostMalloc((void**)&h_D, size, hipHostMallocDefault);
    float* h_F;
    hipHostMalloc((void**)&h_F, size, hipHostMallocDefault);

    int i;
    for (i = 0; i < N; ++i){
        h_A[i] = (float)rand() / RAND_MAX;
        h_B[i] = (float)rand() / RAND_MAX;
        h_D[i] = (float)rand() / RAND_MAX;
    }

    float* d_A;
    hipMalloc((void**)&d_A, size);
    float* d_B;
    hipMalloc((void**)&d_B, size);
    float* d_C;
    hipMalloc((void**)&d_C, size);
    float* d_D;
    hipMalloc((void**)&d_D, size);
    float* d_F;
    hipMalloc((void**)&d_F, size);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&fin);

    hipEventRecord(start, 0);
    hipStream_t stream[2];
    hipStreamCreate(&stream[0]);
    hipStreamCreate(&stream[1]);
    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream[0]);
    VecAdd<<<blocksPerGrid, threadsPerBlock, 0, stream[0]>>>(d_A, d_B, d_C, N);
    hipEventRecord(fin, stream[0]);
    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream[0]);

    hipMemcpyAsync(d_D, h_D, size, hipMemcpyHostToDevice, stream[1]);
    hipStreamWaitEvent(stream[1], fin, 0);
    VecAdd<<<blocksPerGrid, threadsPerBlock, 0, stream[1]>>>(d_C, d_D, d_F, N);
    hipMemcpyAsync(h_F, d_F, size, hipMemcpyDeviceToHost, stream[1]);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&dur_time, start, stop);
    fprintf(stderr, "%.3f\n", dur_time);
    hipEventDestroy(fin);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_D);
    hipFree(d_F);

    freopen(out, "w", stdout);
    for (i = 0; i < N; ++i)
        printf("%.5f %.5f\n", h_C[i], h_F[i]);

    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    hipHostFree(h_D);
    hipHostFree(h_F);

    return 0;
}